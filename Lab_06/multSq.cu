
#include <hip/hip_runtime.h>
#include <stdio.h>


#define SIZE 64  // 64-by-64 square matrix

__global__ void matMult(int * matProd, int * matA, int * matB)
{
        int row = blockIdx.x;
        int col = threadIdx.x;

        int tmpSum = 0;;
        if (row < SIZE && col < SIZE)
        {
                for (int i=0; i<SIZE; ++i)
                        tmpSum += matA[row*SIZE + i] * matB[i*SIZE + col];
                matProd[row*SIZE + col] = tmpSum;
        }
}

int main()
{
        // initialize, aalocate and define host memory
        int matA[SIZE*SIZE] = { 0 };
        int matB[SIZE*SIZE] = { 0 };
        int matProd[SIZE*SIZE] = { 0 };
        for (int i=0; i<SIZE; ++i)
        {
                for (int j=0; j<SIZE; ++j)
                {
                        matA[i*SIZE + j] = i+j;
                        matB[i*SIZE + j] = i-j;
                }
        }

        // initialize and allocate device memory
        int * dev_matProd, * dev_matA, * dev_matB;
        hipMalloc((void **)&dev_matA, SIZE*SIZE*sizeof(int));
        hipMalloc((void **)&dev_matB, SIZE*SIZE*sizeof(int));
        hipMalloc((void **)&dev_matProd, SIZE*SIZE*sizeof(int));


        // copy data to device memory
        hipMemcpy((void *)dev_matA, (void *)matA, SIZE*SIZE*sizeof(int),
                        hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_matB, (void *)matB, SIZE*SIZE*sizeof(int),
                        hipMemcpyHostToDevice);

        matMult<<<SIZE,SIZE>>>(dev_matProd, dev_matA, dev_matB);
        
        // check for successful thread execution
        if (hipDeviceSynchronize() != hipSuccess)
        {
                printf("Error\n");
                return -1;
        }

        // copy results from device to host memory
        hipMemcpy(matProd, dev_matProd, SIZE*SIZE*sizeof(int),
                        hipMemcpyDeviceToHost);


        for (int i=0; i<SIZE/2; ++i)  // inspecting first few diagnols
                printf(" > Diagonal %d of prudect is %d.\n",
                                i, matProd[i*SIZE+i]);

        return 0;
}
