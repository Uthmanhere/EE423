
#include <hip/hip_runtime.h>
#define N 10000

__global__ void compute( float * a, float * b, float * c )
{
        int tid = blockIdx.x;
        if (tid < N)
                c[tid] = b[tid] / a[tid];
}

int main(void)
{
        float a[N], b[N], c[N];
        float * dev_a, * dev_b, * dev_c;

        hipMalloc((void **)&dev_a, N*sizeof(int));
        hipMalloc((void **)&dev_b, N*sizeof(int));
        hipMalloc((void **)&dev_c, N*sizeof(int));

        for (int i=0; i<N; i++)
        {
                a[i] = cos(i);
                b[i] = sin(i);
        }

        hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

        compute<<<N,1>>>(dev_a, dev_b, dev_c);

        hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

        for (int i=0; i<10; i++)
                printf(">> for i %d ocomputers %f.\n", i, c[i]);

        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);

        return 0;
}
