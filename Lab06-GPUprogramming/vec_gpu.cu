
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10000  // vector size


// kernel function for device
__global__ void compute( float * a, float * b, float * c )
{
        // index by block ID
        int i = blockIdx.x;
        if (i < N)  // check if thread is valid
                c[i] = b[i] / a[i];  // compute tangent
}

int main(void)
{
        // initialize and define host vectors
        float a[N], b[N], c[N];
        
        for (int i=0; i<N; i++)
        {
                a[i] = cos(i);
                b[i] = sin(i);
        }   
        // initialize and allocate device pointers
        float * dev_a, * dev_b, * dev_c;

        hipMalloc((void **)&dev_a, N*sizeof(float));
        hipMalloc((void **)&dev_b, N*sizeof(float));
        hipMalloc((void **)&dev_c, N*sizeof(float));

        // copy data to host memory
        hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

        // call kernel for N bloack, 1 thread each
        compute<<<N,1>>>(dev_a, dev_b, dev_c);
      
        //check for successful thread execution
        if (hipDeviceSynchronize() != hipSuccess)
        {
                printf("Error\n");
                return -1;
        }

        // copy results back to host memory
        hipMemcpy(c, dev_c, N*sizeof(float), hipMemcpyDeviceToHost);

        // print a few results for observation
        for (int i=0; i<10; i++)
                printf(">> for i %d ocomputes %f.\n", i, c[i]);

        // free device memory
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);

        return 0;
}
